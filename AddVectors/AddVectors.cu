
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void addVectors(int *a, int *b, int *c, int n) {
	int thread = threadIdx.x;

	if(thread < n)
		c[thread] = a[thread] + b[thread];
}

int main() {
	int *a = NULL;
	int *b = NULL;
	int *c = NULL;
	int *dev_a = NULL;
	int *dev_b = NULL;
	int *dev_c = NULL;
	int size = 10;


	a = (int *) malloc(sizeof(int) * size);
	b = (int *) malloc(sizeof(int) * size);
	c = (int *) malloc(sizeof(int) * size);

	for(int i = 0; i < size; i++) {
		a[i] = i;
		b[i] = i;
	}

	hipMalloc(&dev_a, size * sizeof(int));
	hipMalloc(&dev_b, size * sizeof(int));
	hipMalloc(&dev_c, size * sizeof(int));

	hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c, size * sizeof(int), hipMemcpyHostToDevice);

	addVectors<<<1, 1024>>>(dev_a, dev_b, dev_c, size);

	hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

	printf("Your result vector is: \n");
	for(int i = 0; i < size; i++)
		printf("c[%d] = %d\n", i, c[i]);


	free(a);
	free(b);
	free(c);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}
